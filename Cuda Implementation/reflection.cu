#pragma once

#include "hip/hip_runtime.h"


#include <iostream>

////////////////////////////////////////////////////////////////////////

using namespace std;

////////////////////////////////////////////////////////////////////////

template <typename Type>

struct Reflection
{
	Type* host = nullptr;
	Type* device = nullptr;

	unsigned int size = 0;
};

////////////////////////////////////////////////////////////////////////

template <typename Type>

Reflection<Type> Malloc(const unsigned int count)
{
	const unsigned int size = count * sizeof(Type);

	Reflection<Type> reflection;

	if(size == 0)
	{
		return reflection;
	}

	if(hipMalloc(&reflection.device, size) != hipSuccess)
	{
		reflection.device = nullptr;
		return reflection;
	}

	if(hipMemset(reflection.device, 0, size) != hipSuccess)
	{
		hipFree(reflection.device);
		reflection.device = nullptr;
		return reflection;
	}

	reflection.host = new Type[count];

	memset(reflection.host, 0, size);

	reflection.size = size;

	return reflection;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

Reflection<Type> Malloc(Type* hostBuffer, const unsigned int count, bool send = false)
{
	const unsigned int size = count * sizeof(Type);

	Reflection<Type> reflection;

	if(size == 0)
	{
		return reflection;
	}

	if(hipMalloc(&reflection.device, size) != hipSuccess)
	{
		reflection.device = nullptr;
		return reflection;
	}

	if(!send)
	{
		if(hipMemset(reflection.device, 0, size) != hipSuccess)
		{
			hipFree(reflection.device);
			reflection.device = nullptr;
			return reflection;
		}
	}

	reflection.host = new Type[count];

	memcpy(reflection.host, hostBuffer, size);

	reflection.size = size;

	if(send)
	{
		Send(reflection);
	}

	return reflection;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

void Free(Reflection<Type>& reflection)
{
	if(reflection.size)
	{
		if(reflection.host != nullptr)
		{
			delete []reflection.host;
			reflection.host = nullptr;
		}

		if(reflection.device != nullptr)
		{
			hipFree(reflection.device);
			reflection.device = nullptr;
		}

		reflection.size = 0;
	}
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

bool IsValid(Reflection<Type>& reflection)
{
	if(reflection.size == 0 || reflection.host == nullptr || reflection.device == nullptr)
	{
		return false;
	}

	return true;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

bool Send(Reflection<Type>& reflection)
{
	if(!IsValid(reflection))
	{
		return false;
	}

	return hipMemcpy(reflection.device, reflection.host, reflection.size, hipMemcpyHostToDevice) == hipSuccess;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

bool Send(Reflection<Type>& reflection, const unsigned int count)
{
	if(!IsValid(reflection))
	{
		return false;
	}

	if(count == 0)
	{
		return true;
	}

	unsigned int size = count * sizeof(Type);

	if(size > reflection.size)
	{
		size = reflection.size;

		throw "Invalid Argument Exeption";
	}

	return hipMemcpy(reflection.device, reflection.host, size, hipMemcpyHostToDevice) == hipSuccess;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

bool Receive(Reflection<Type>& reflection)
{
	if(!IsValid(reflection))
	{
		return false;
	}

	return hipMemcpy(reflection.host, reflection.device, reflection.size, hipMemcpyDeviceToHost) == hipSuccess;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

bool Receive(Reflection<Type>& reflection, const unsigned int count)
{
	if(!IsValid(reflection))
	{
		return false;
	}

	if(count == 0)
	{
		return true;
	}

	unsigned int size = count * sizeof(Type);

	if(size > reflection.size)
	{
		size = reflection.size;

		throw "Invalid Argument Exeption";
	}

	return hipMemcpy(reflection.host, reflection.device, size, hipMemcpyDeviceToHost) == hipSuccess;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

Type* Host(Reflection<Type>& reflection)
{
	return reflection.host;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

Type* Device(Reflection<Type>& reflection)
{
	return reflection.device;
}

////////////////////////////////////////////////////////////////////////

template <typename Type>

void Clear(Reflection<Type>& reflection)
{
	if(reflection.size)
	{
		if(reflection.host != nullptr)
		{
			memset(reflection.host, 0, reflection.size);
		}
	}
}

////////////////////////////////////////////////////////////////////////





